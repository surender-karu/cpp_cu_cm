#include "hip/hip_runtime.h"
#include "cuda_common.cuh"

__global__ void matrixAdd(float *A, float *B, float *C, int n) {
  int tid = threadIdx.x;
  int blkIdx = blockIdx.x;

  C[tid * n + blkIdx] = A[tid * n + blkIdx] + B[tid * n + blkIdx];

  // for(int i = 0; i < n; i++) {
  //   for(int j = 0; j < n; j++) {
  //     C[i * n + j] = A[i * n + j] + B[i * n + j];
  //   }
  // }
}

void matrixAdd(float *A, float *B, float *C, int n, int grid_size,
               int block_size) {
  float *d_A, *d_B, *d_C;

  size_t matrix_size = n * n * sizeof(float);

  hipMalloc(reinterpret_cast<void **>(&d_A), matrix_size);
  hipMalloc(reinterpret_cast<void **>(&d_B), matrix_size);
  hipMalloc(reinterpret_cast<void **>(&d_C), matrix_size);

  hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, n * n * sizeof(float), hipMemcpyHostToDevice);

  matrixAdd<<<n, n>>>(d_A, d_B, d_C, n);

  hipMemcpy(C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}